#include "hip/hip_runtime.h"
// Name: Brooks Pearce
// Simple Julia CPU.
// nvcc HW6.cu -o temp -lglut -lGL
// nvcc HW7.cu -o temp -lglut -lGLU -lGL -lm
// glut and GL are openGL libraries.
/*
 What to do:
 This code displays a simple Julia fractal using the CPU.
 Rewrite the code so that it uses the GPU to create the fractal. 
 Keep the window at 1024 by 1024.
*/

// Include files
#include <stdio.h>
#include <GL/glut.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <GL/gl.h> 

// Defines
#define MAXMAG 10.0 // If you grow larger than this, we assume that you have escaped.
#define MAXITERATIONS 200 // If you have not escaped after this many attempts, we assume you are not going to escape.
//#define A  -0.824	//Real part of C
//#define B  -0.1711	//Imaginary part of C

// Global variables
unsigned int WindowWidth = 1024;
unsigned int WindowHeight = 1024;
float *HostPixels;
float *DevicePixels;
double aStep,bStep;
const float A = -0.824;
const floT B = -0.1711;

float XMin = -2.0;
float XMax =  2.0;
float YMin = -2.0;
float YMax =  2.0;

dim3 GridSize;
dim3 BlockSize;

// Function prototypes
void cudaErrorCheck(const char*, int);
__device__ float escapeOrNotColor(float, float, double, double);
__global__ void kernel(float* pixels,float XMin,float XMax,float YMin,float YMax,int WindowHeight,int WindowWidth,double aStep,double bStep);
void display(void);	
void animate(void);
void freeMemory();

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

__device__ float escapeOrNotColor(float x, float y, double aStep, double bStep) 
{
	float mag,tempX;
	int count;
	
	int maxCount = MAXITERATIONS;
	float maxMag = MAXMAG;
	
	count = 0;
	mag = sqrtf(x*x + y*y);;
	while (mag < maxMag && count < maxCount) 
	{	
		tempX = x; //We will be changing the x but we need its old value to find y.
		x = x*x - y*y + aStep;
		y = (2.0 * tempX * y) + bStep;
		mag = sqrtf(x*x + y*y);
		count++;
	}
	if(count < maxCount) 
	{
		return(0.0);
	}
	else
	{
		return(1.0);
	}
}

__global__ void kernel(float *pixels,float XMin,float XMax,float YMin,float YMax,int WindowHeight,int WindowWidth,double aStep, double bStep) 	
{
	    
		int ix = blockIdx.x * blockDim.x + threadIdx.x;
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		if (ix >= WindowWidth || iy >= WindowHeight) return;
		
		float stepSizeX = (XMax - XMin)/((float)WindowWidth);
		float stepSizeY = (YMax - YMin)/((float)WindowHeight);
		
		
		int idx = (iy * WindowWidth + ix) * 3; 
		float x = stepSizeX * ix +  XMin;
		float y = stepSizeY * iy + YMin;
		
		if (y < YMax && x < XMax)
		{
			pixels[idx] = escapeOrNotColor(x,y,aStep,bStep);	
			pixels[idx+1] = 0.0; 
			pixels[idx+2] = 0.0;		
			
		}
		return; 
}
void initGL() {
    glDisable(GL_DEPTH_TEST);
    glClearColor(0.f, 0.f, 0.f, 1.f);
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);  // important for tightly packed float RGB

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    // Ortho so pixel coords map 1:1 to window
    gluOrtho2D(0, WindowWidth, 0, WindowHeight);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glViewport(0, 0, WindowWidth, WindowHeight);
}

/*void reshape(int w, int h){
    WindowWidth = w; WindowHeight = h;
    glViewport(0,0,w,h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, w, 0, h);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}*/

void display(void) 
{ 
	// Set Block & Grid Dim
	BlockSize.x = 16;
	BlockSize.y = 16;
	BlockSize.z = 1;
	GridSize.x = (WindowWidth  + BlockSize.x - 1) / BlockSize.x;
	GridSize.y = (WindowHeight  + BlockSize.y - 1) / BlockSize.y;
	GridSize.z = 1;
	
	kernel<<<GridSize, BlockSize>>>(DevicePixels,XMin,XMax,YMin,YMax,WindowHeight,WindowWidth,aStep,bStep);
	
	cudaErrorCheck(__FILE__, __LINE__);
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipMemcpy(HostPixels, DevicePixels, WindowWidth*WindowHeight*3*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	glClear(GL_COLOR_BUFFER_BIT);
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1)
    glWindowPos2i(0, 0); 
    
	
	//Putting pixels on the screen.
	glDrawPixels(WindowWidth, WindowHeight, GL_RGB, GL_FLOAT, HostPixels); 
	//glFlush(); 
	glutSwapBuffers();
}
void animate(void)
{
	const int radius = 0.5;
	const int omega = 0.5;
	float t = 0.001f * (float)glutGet(GLUT_ELAPSED_TIME);
	aStep = A + radius*cosf(omega*t);
	bStep = B + radius*sinf(omega*t);
	
	glutPostRedisplay();
}
void freeMemory() {
    if (DevicePixels) hipFree(DevicePixels);
    if (HostPixels) free(HostPixels);
}


int main(int argc, char** argv)
{ 
	aStep = A;
	bStep = B ;
	
	HostPixels = (float *)malloc(WindowWidth*WindowHeight*3*sizeof(float));
	hipMalloc(&DevicePixels, WindowWidth*WindowHeight*3*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	atexit(freeMemory);

   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
   	glutInitWindowSize(WindowWidth, WindowHeight);
	glutCreateWindow("Fractals--Man--Fractals");
	initGL();
	//glutReshapeFunc(reshape);
   	glutDisplayFunc(display);
	glutIdleFunc(animate);

	
   	glutMainLoop(); 
}

