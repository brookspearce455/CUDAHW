#include "hip/hip_runtime.h"
// Name: Brooks Pearce
// Simple Julia CPU.
// nvcc HW6.cu -o temp -lglut -lGL
// glut and GL are openGL libraries.
/*
 What to do:
 This code displays a simple Julia fractal using the CPU.
 Rewrite the code so that it uses the GPU to create the fractal. 
 Keep the window at 1024 by 1024.
*/

// Include files
#include <stdio.h>
#include <GL/glut.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Defines
#define MAXMAG 10.0 // If you grow larger than this, we assume that you have escaped.
#define MAXITERATIONS 200 // If you have not escaped after this many attempts, we assume you are not going to escape.
//#define A  -0.824	//Real part of C
//#define B  -0.1711	//Imaginary part of C

// Global variables
unsigned int WindowWidth = 1024;
unsigned int WindowHeight = 1024;
float *HostPixels;
float *DevicePixels;

float XMin = -2.0;
float XMax =  2.0;
float YMin = -2.0;
float YMax =  2.0;

dim3 GridSize;
dim3 BlockSize;

// Function prototypes
void cudaErrorCheck(const char*, int);
float escapeOrNotColor(float, float);
__global__ void kernel(float* pixels,float XMin,float XMax,float YMin,float YMax,int WindowHeight,int WindowWidth,float A,float B);
void display(void);	
void freeMemory();

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

__device__ float escapeOrNotColor (float x, float y, float A, float B) 
{
	float mag,tempX;
	int count;
	
	int maxCount = MAXITERATIONS;
	float maxMag = MAXMAG;
	
	count = 0;
	mag = sqrt(x*x + y*y);;
	while (mag < maxMag && count < maxCount) 
	{	
		tempX = x; //We will be changing the x but we need its old value to find y.
		x = x*x - y*y + A;
		y = (2.0 * tempX * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	if(count < maxCount) 
	{
		return(0.0);
	}
	else
	{
		return(1.0);
	}
}

__global__ void kernel(float *pixels,float XMin,float XMax,float YMin,float YMax,int WindowHeight,int WindowWidth,float A, float B) 	
{
	    
		int ix = blockIdx.x * blockDim.x + threadIdx.x;
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		float stepSizeX = (XMax - XMin)/((float)WindowWidth);
		float stepSizeY = (YMax - YMin)/((float)WindowHeight);
		
		int idx = (iy * WindowWidth + ix) * 3; 
		float x = stepSizeX * ix +  XMin;
		float y = stepSizeY * iy + YMin;
		
		if (y < YMax && x < XMax)
		{
			pixels[idx] = escapeOrNotColor(x,y,A,B);	
			pixels[idx+1] = 0.0; 
			pixels[idx+2] = 0.0;		
			
		}
		return; 
}
void display(void) 
{ 
	// Set Block & Grid Dim
	BlockSize.x = 16;
	BlockSize.y = 16;
	BlockSize.z = 1;
	GridSize.x = (WindowWidth  + BlockSize.x - 1) / BlockSize.x;
	GridSize.y = (WindowHeight  + BlockSize.y - 1) / BlockSize.y;
	GridSize.z = 1;
	
	const float A = -0.824;	//Real part of C
	const float B = -0.1711;	//Imaginary part of C
	const float omega = 0.5;
	const float radius = 0.5;
	
	float A1 = A + radius*cos(omega*t);
	float B1 = B + radius*sin(omega*t);
	kernel<<<GridSize, BlockSize>>>(DevicePixels,XMin,XMax,YMin,YMax,WindowHeight,WindowWidth,A,B);
	
	cudaErrorCheck(__FILE__, __LINE__);
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipMemcpy(HostPixels, DevicePixels, WindowWidth*WindowHeight*3*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Putting pixels on the screen.
	glDrawPixels(WindowWidth, WindowHeight, GL_RGB, GL_FLOAT, HostPixels); 
	glFlush(); 
}
void idle(void)
{
	glutPostRedisplay();
}
void freeMemory() {
    if (DevicePixels) hipFree(DevicePixels);
    if (HostPixels) free(HostPixels);
}


int main(int argc, char** argv)
{ 
	HostPixels = (float *)malloc(WindowWidth*WindowHeight*3*sizeof(float));
	hipMalloc(&DevicePixels, WindowWidth*WindowHeight*3*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	freeMemory(onExit);

   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
   	glutInitWindowSize(WindowWidth, WindowHeight);
	glutCreateWindow("Fractals--Man--Fractals");
   	glutDisplayFunc(display);
	glutIdleFunction(idle);
   	glutMainLoop();
}

