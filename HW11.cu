

// Name: Brooks Pearce
// Robust Vector Dot product 
// nvcc HW11.cu -o temp

// Include files
#include <sys/time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// Defines
#define N 2000000 // Length of the vector
#define BLOCK_SIZE 1024 // Threads in a block

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;



// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void dotProductCPU(float*, float*, int);
__global__ void dotProductGPU(float*, float*, double*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();
void selectDevice();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	// This code can stride the length of the grid so block count doesn't become a bottleneck
	GridSize.x = 10000; // This gives us the correct number of blocks.
	GridSize.y = 1;
	GridSize.z = 1;
	
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Padding with zeros 
	hipMemset(A_GPU, 0, N*sizeof(float)); 
	hipMemset(B_GPU, 0, N*sizeof(float));
	hipMemset(C_GPU, 0, sizeof(float));
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(3*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	for(int id = 1; id < n; id++)
	{ 
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *c, int n)
{
__shared__ float cache[BLOCK_SIZE];
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int cacheIndex = threadIdx.x;
	float sum = 0;

	 // Zeroing out the cache for every block just in case
	cache[cacheIndex] = 0;
	__syncthreads();
	
	while (id < n)
	{	
		sum += a[id] * b[id];
		cache[cacheIndex] = sum;
		id += blockDim.x * gridDim.x;
		
	}
	__syncthreads();	
	
	int fold = blockDim.x/2;
	while (fold > 0) 
	{
		if (cacheIndex < fold) 
		{
			cache[cacheIndex] +=  cache[cacheIndex + fold];
		}
		fold /=2;
		__syncthreads();
	}
	
	// Atomic add adds the first element in the cache for each block to the first element in c
	if (threadIdx.x == 0)
	{
		
		__syncthreads();
		atomicAdd(&c[0], cache[0]);
		
	}
	
}

// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{	
	
	double percentError;
	
	percentError = fabs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

// Uses the device prop to iterate through the devices and chooses the one with the highest compute capability
void selectDevice()
{
	int count;
	int bestCompute = 0;
	int bestGPU;
	int gridSize = (N-1)/BLOCK_SIZE+1; // calculates number of blocks because setUpDevices(); hasn't been called yet
	
	hipDeviceProp_t prop;
	hipGetDeviceCount(&count);
	cudaErrorCheck(__FILE__, __LINE__);
	
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop,i);
		cudaErrorCheck(__FILE__, __LINE__);
		printf(" ---General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Total global mem: %ld\n\n", prop.totalGlobalMem);
		if (prop.major > bestCompute)
		{
			bestCompute = prop.major;
			bestGPU = i;
		}
	}
	printf("Chosen GPU is: device %d\n\n",bestGPU);
	hipSetDevice(bestGPU);
	if (prop.maxThreadsPerBlock < BLOCK_SIZE || prop.maxGridSize[0] < gridSize)
	{
		printf("Thread count or block count didn't meet code expectations. Go get a new GPU!\n");
		exit(0);
	}
	
}

double percentVramUsed ()
{
	double percentUsed;
	size_t freeMem;
	size_t totalMem;
	
	hipMemGetInfo(&freeMem,&totalMem);
	
	
	percentUsed = 100 - (freeMem * 100 / totalMem);
	return(percentUsed);
}

int main()
{	
	double percentUsed;
	timeval start, end;
	long timeCPU, timeGPU;
	//float localC_CPU, localC_GPU;
	
	// Selecting the GPU
	selectDevice();
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, sizeof(float), hipMemcpyDeviceToHost); // only copying over the first element in the GPU vector
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Making sure the GPU and CPU wait until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	DotGPU = C_CPU[0];
	printf("DotCPU: %f\nDotGPU: %f\n", DotCPU, DotGPU);

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Compares free and total memory on the GPU
	percentUsed = percentVramUsed();
	printf("\n Percentage of VRAM used: %lf\n",percentUsed);
	
	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}





